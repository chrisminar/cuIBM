#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file projectVelocity.cu
 * \author Anush Krishnan (anush@bu.edu),
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief kernels to update the velocity field
 */


#include "projectVelocity.h"

namespace kernels 
{
__global__
void project_velocity_luo_X(double *u, double *uhat, double *uold, double *pressure, double *dx, double dt, int nx, int ny)
{
	int i	= threadIdx.x + (blockDim.x * blockIdx.x),
		I	= i % (nx-1),
		J 	= i / (nx-1),
		ip  = J*nx + I,
		numU= (nx-1)*ny;

	if (i >= numU)
		return;

	uold[i] = u[i];

	u[i] = uhat[i] - dt*(pressure[ip+1]-pressure[ip]) / (0.5*dx[I+1]+0.5*dx[I]);
}

__global__
void project_velocity_luo_Y(double *u, double *uhat, double *uold, double *pressure, double *dy, double dt, int nx, int ny)
{
	int numU= (nx-1)*ny,
		i	= threadIdx.x + (blockDim.x * blockIdx.x),
		I	= i % nx,
		J	= i / nx,
		ip	= J*nx + I,
		numUV	= (ny-1)*nx + numU;

	i += numU;

	if (i >= numUV)
		return;

	uold[i] = u[i];

	u[i] = uhat[i] - dt*(pressure[ip+nx]-pressure[ip]) / (0.5*dy[J+1]+0.5*dy[J]);
}

}//end namespace kernels
