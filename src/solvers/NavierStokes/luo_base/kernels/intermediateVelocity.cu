#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file intermediateVelocity.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief kernels to generate the right hand side for the initial velocity solve
 */


#include "intermediateVelocity.h"

/**
 * \namespace kernels
 * \brief Contains all the custom-written CUDA kernels.
 */

namespace kernels
{
__global__
void setInsideVelocity(int *ghostTags, double *u, double *uB, double *vB, int nx, int ny) //flag doesn't need to cover whole domain, could only span over the bounding box
{																  //flag kernel could mess up if the body is too close to the edge because were doing the x values and y values in the same kernel
	int 	i 	= threadIdx.x + (blockDim.x * blockIdx.x),
			I	= i % (nx-1),
			J	= i / (nx-1),
			iu	= J*(nx-1) + I,
			iv	= J*nx + I + (nx-1)*ny;

	if (iu >= (nx-1)*ny) //flag indexing is janky for doing x and y at the same time
			return;
	//			 not at inside edge             at inside edge
	u[iu] = (ghostTags[iu] != 0) * u[iu] + (ghostTags[iu] == 0) * uB[0];//flag won't work for rotating bodies because were not getting a local body velocity
	u[iv] = (ghostTags[iv] != 0) * u[iv] + (ghostTags[iv] == 0) * vB[0];
}
}
