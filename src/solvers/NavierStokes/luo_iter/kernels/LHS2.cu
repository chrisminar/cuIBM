#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file LHS2.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief kernels to generate the left hand side for the poission solve
 */

#include "LHS2.h"

namespace kernels
{
__global__
void LHS2_mid_iter(int *row, int *col, double *val, double *dx, double *dy, int nx, int ny, double dt,
					int *count, double *ns_rhs, double *interp_rhs, int *hybridTagsP, int *ghostTagsP,
					double *alpha, double *dpdn,
					int *index1, int *index2, int *index3, int *index4,
					double *q1coef, double *q2coef, double *q3coef, double *q4coef,
					double *q1, double *q2, double *q3, double *q4, int timeStep)
{
	int ip 	= threadIdx.x + blockDim.x * blockIdx.x;
	if (ip >= nx*ny)
		return;
	int	I	= ip % nx,
		J	= ip / nx;

	if (I == 0 || I == nx-1 || J == 0 || J == ny-1)
		return;

	int numE = nx*4-2 + (J-1)*(nx*5-2) + I*5-1;
	double temp = 0;


	if (hybridTagsP[ip]>0)//if were at hybrid node
	{
		int interp_index[4] = {index1[ip], index2[ip], index3[ip], index4[ip]};
		//int nx_index[5] = {ip+nx, ip+1, ip-nx, ip-1, ip};//n e s w p
		double CInterp[4];
		double Cns[5];
		double q[4] = {q1[ip], q2[ip], q3[ip], q4[ip]};

		//calculate the pressure coefficients for the stencil pressure calculation
		Cns[0] = -dt/(dy[J]*(dy[J]+dy[J+1])*0.5); //N
		Cns[1] = -dt/(dx[I]*(dx[I]+dx[I+1])*0.5); //E
		Cns[2] = -dt/(dy[J]*(dy[J]+dy[J-1])*0.5); //s
		Cns[3] = -dt/(dx[I]*(dx[I]+dx[I-1])*0.5); //w
		Cns[4] = -Cns[0] - Cns[1] - Cns[2] - Cns[3]; //P

		//set pressure coefficients for the interpolation pressure calculation
		CInterp[0] = q1coef[ip];
		CInterp[1] = q2coef[ip];
		CInterp[2] = q3coef[ip];
		CInterp[3] = q4coef[ip];

		//multiply by alpha
		for (int i=0;i<4;i++)
		{
			Cns[i] = (1-alpha[ip])*Cns[i]/Cns[4];
			CInterp[i] = alpha[ip]*CInterp[i];
		}

		/*   0  1  2		NW  N   NE
		 *   3  4  5		W   P   E
		 *   6  7  8		SW  S   SE
		 */
		int stencil_index[9]    = {ip + nx - 1, ip + nx, ip + nx + 1,
								   ip - 1     , ip     , ip + 1,
								   ip - nx - 1, ip - nx, ip - nx + 1};
		double stencil[9] = {0, Cns[0], 0, Cns[3], 1, Cns[1], 0, Cns[2], 0};
		//combine ns and interp stencils
		bool stencil_used[9] = {false, true, false, true, true, true, false, true, false};
		for (int n=0;n<4;n++)
		{
			for (int m=0;m<9;m++)
			{
				if (stencil_index[m] == interp_index[n] && m != 4)
				{
					stencil[m] -= CInterp[n]; //this should be minus
				}
			}
		}
		//add ns to sparse matrix
		for (int m = 0; m<9; m++)
		{
			if (stencil_used[m])
			{
				row[numE] = ip;
				col[numE] = stencil_index[m];
				val[numE] = stencil[m];
				numE++;
			}
		}
		ns_rhs[ip] = (1-alpha[ip])/Cns[4];
		interp_rhs[ip] = 0;
		//calc new numE
		numE = ny*nx*5 - ny*2 - nx*2 + count[ip]-1;
		//add interp corner to sparse matrix
		for (int n=0;n<4;n++)
		{
			for (int m=0;m<9;m++)
			{
				if (stencil_index[m] == interp_index[n] && !stencil_used[m])
				{
					row[numE] = ip;
					col[numE] = interp_index[n];
					val[numE] = -CInterp[n];//this should also be minus
				}
				else if(stencil_index[m] == ip && interp_index[n] == ip)
					interp_rhs[ip] += CInterp[n]*q[n]; //this should be addition

			}
		}
	}
	else if (ghostTagsP[ip]>0)
	{
		int interp_index[4] = {index1[ip], index2[ip], index3[ip], index4[ip]};
		bool interp_in[4] = {false, false, false, false};
		int ns_index[5] = {ip + nx, ip + 1, ip - nx, ip -1, ip}; //n e s w p
		bool ns_overlap[5] = {false, false, false, false, true};
		double q[4] = {q1[ip], q2[ip], q3[ip], q4[ip]};
		double CInterp[4];
		CInterp[0] = q1coef[ip];
		CInterp[1] = q2coef[ip];
		CInterp[2] = q3coef[ip];
		CInterp[3] = q4coef[ip];
		//count the number of nodes the interp is using
		//find how which ns nodes are occupied
		int counter = 0;
		temp = 0;
		for (int l=0; l<4; l++)
		{
			if (ghostTagsP[interp_index[l]]>0)
			{
				counter +=1;
				interp_in[l] = true;
			}
			for (int n=0; n<5; n++)
			{
				if (interp_index[l] == ns_index[n])
					ns_overlap[n] = true;
			}
		}
		//add center to matrix
		row[numE] = ip;
		col[numE] = ip;
		val[numE] = 1;
		numE++;
		//add real interp values to matrix
		for (int i=0; i<4; i++)
		{
			if (!interp_in[i] && interp_index[i] != ip)
			{
				row[numE] = ip;
				col[numE] = interp_index[i];
				val[numE] = -CInterp[i];
				numE++;
			}
			else
			{
				temp += CInterp[i] * q[i];
			}
		}
		//fill remainder of values
		int counter2 = 0;
		for (int i=0; i<5; i++)
		{
			if (counter2>=counter)
				break;
			if (ns_overlap[i]==false)
			{
				row[numE] = ip;
				col[numE] = ns_index[i];
				val[numE] = 0;
				numE++;
				counter2++;
			}
		}
		ns_rhs[ip] = 0;
		interp_rhs[ip] = dpdn[ip] + temp;
	}
	else //if were not at a hybrid node
	{
		//temp = dt/(dx[I]*(dx[I]+dx[I+1])*0.5) + dt/(dx[I]*(dx[I]+dx[I-1])*0.5) + dt/(dy[J]*(dy[J]+dy[J+1])*0.5) + dt/(dy[J]*(dy[J]+dy[J-1])*0.5);
		temp = 0;
		//EAST
		row[numE] = ip;
		col[numE] = ip + 1;
		//val[numE] = -dt/(dx[I]*(dx[I]+dx[I+1])*0.5)/temp;
		val[numE] = -dt/(dx[I]*(dx[I]+dx[I+1])*0.5);
		temp += dt/(dx[I]*(dx[I]+dx[I+1])*0.5);
		numE++;

		//WEST
		row[numE] = ip;
		col[numE] = ip - 1;
		//val[numE] = -dt/(dx[I]*(dx[I]+dx[I-1])*0.5)/temp;
		val[numE] = -dt/(dx[I]*(dx[I]+dx[I-1])*0.5);
		temp +=dt/(dx[I]*(dx[I]+dx[I-1])*0.5);
		numE++;

		//NORTH
		row[numE] = ip;
		col[numE] = ip + nx;
		//val[numE] = -dt/(dy[J]*(dy[J]+dy[J+1])*0.5)/temp;
		val[numE] = -dt/(dy[J]*(dy[J]+dy[J+1])*0.5);
		temp += dt/(dy[J]*(dy[J]+dy[J+1])*0.5);
		numE++;

		//SOUTH
		row[numE] = ip;
		col[numE] = ip - nx;
		//val[numE] = -dt/(dy[J]*(dy[J]+dy[J-1])*0.5)/temp;
		val[numE] = -dt/(dy[J]*(dy[J]+dy[J-1])*0.5);
		temp += dt/(dy[J]*(dy[J]+dy[J-1])*0.5);
		numE++;
		//MID
		row[numE] = ip;
		col[numE] = ip;
		//val[numE] = 1;
		val[numE] = temp;
		ns_rhs[ip] = 1;
		interp_rhs[ip] = 0;
		//ns_rhs[ip] = 1/temp;
		//interp_rhs[ip] = 0;
	}
}
}
